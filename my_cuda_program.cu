#include <stdio.h>
#include <hip/hip_runtime.h>  // ต้องใช้ไลบรารีนี้สำหรับฟังก์ชัน CUDA

// CUDA kernel function
__global__ void helloCUDA() {
    printf("Hello from GPU!\n");
}

int main() {
    // เรียกใช้ kernel ด้วย 1 block และ 1 thread
    helloCUDA<<<1, 1>>>();

    // ซิงค์ CPU กับ GPU เพื่อรอให้ GPU ทำงานเสร็จ
    hipDeviceSynchronize();

    return 0;
}
