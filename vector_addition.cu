#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA Kernel function to add elements of two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 5;
    int a[] = {1, 2, 3, 4, 5};
    int b[] = {10, 20, 30, 40, 50};
    int c[n];

    // Allocate memory on the GPU
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    // Copy data from CPU to GPU
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel on the GPU
    add<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy the result back to the CPU
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free the GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print the result
    printf("Result: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    return 0;
}
