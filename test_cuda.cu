
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

int main() {
    // Launch a kernel with 1 block and 1 thread
    helloFromGPU<<<1, 1>>>();

    // Synchronize the device to ensure the kernel is complete
    hipDeviceSynchronize();

    return 0;
}
